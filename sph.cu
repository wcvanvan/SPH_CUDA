#include <iostream>
#include "sph.h"

Particle *initParticles(int particleCount)
{
    Particle *particles;
    hipMallocManaged(&particles, particleCount * sizeof(Particle));
    return particles;
}

void updateSimulation(Particle *particles, int particleCount)
{
    int blockDim = 32;
    int gridDim = (particleCount + (blockDim - 1)) / blockDim;
    hipError_t err;
    // TODO: call computation kernels here
	// hipDeviceSynchronize();

    if ((err = hipGetLastError()) != hipSuccess)
    {
        std::cerr << "Kernel error: " << hipGetErrorString(err) << std::endl;
    }
}
