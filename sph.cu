#include <iostream>
#include "sph.h"

Particle *SPHInit()
{
    Particle *particles;
    hipMallocManaged(&particles, MAX_PARTICLES * sizeof(Particle));
    return particles;
}

void updateSimulation(Particle *particles, int particleCount)
{
    int blockDim = 32;
    int gridDim = (particleCount + (blockDim - 1)) / blockDim;
    hipError_t err;
    // TODO: call computation kernels here
	// hipDeviceSynchronize();

    if ((err = hipGetLastError()) != hipSuccess)
    {
        std::cerr << "Kernel error: " << hipGetErrorString(err) << std::endl;
    }
}
