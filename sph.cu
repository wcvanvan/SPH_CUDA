#include "hip/hip_runtime.h"
#include <iostream>
#include "sph.h"

uint32_t *gpuAlloc()
{
    uint32_t *host_ptr;

    hipError_t err = hipHostAlloc(&host_ptr, SCREEN_SIZE * sizeof(uint32_t), hipHostMallocMapped);
    if (err != hipSuccess)
    {
        std::cerr << "hipHostAlloc Error " << hipGetErrorString(err) << std::endl;
    }
    return host_ptr;
};

void gpuFree(void *host_ptr)
{
    hipHostFree(host_ptr);
}

__global__ void fillScreenKernel(uint32_t *buf)
{
    const int pixelX = blockDim.x * blockIdx.x + threadIdx.x;
    const int pixelY = blockDim.y * blockIdx.y + threadIdx.y;
    if (pixelX >= SCREEN_WIDTH || pixelY >= SCREEN_HEIGHT)
    {
        return;
    }
    unsigned int pos = SCREEN_WIDTH * pixelY + pixelX;
    buf[pos] = 0xFFFF0000;
}

void SPHSimulation(uint32_t *host_ptr)
{
    uint32_t *device_ptr;
    hipError_t err = hipHostGetDevicePointer(&device_ptr, host_ptr, 0);
    if (err != hipSuccess)
    {
        std::cerr << "hipHostGetDevicePointer error: " << hipGetErrorString(err) << std::endl;
        return;
    }
    const dim3 gridDim(H_TILES, V_TILES);
    const dim3 blockDim(TILE_WIDTH, TILE_HEIGHT);
    fillScreenKernel<<<gridDim, blockDim>>>(device_ptr);

    if ((err = hipGetLastError()) != hipSuccess)
    {
        std::cerr << "Kernel error: " << hipGetErrorString(err) << std::endl;
    }
    hipDeviceSynchronize();
}
