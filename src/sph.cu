#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include "const.h"
#include "sph.h"

// Assign a cell id to each particle based on its position.
__global__ void computeCellId(Particle *particles, int particlesCount, float cellSize, float xLen, float yLen,
                              float zLen, int gridDimX, int gridDimY, int gridDimZ) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= particlesCount) return;
  Particle &particle = particles[i];

  float halfX = xLen / 2.0f;
  float halfY = yLen / 2.0f;
  float halfZ = zLen / 2.0f;
  int ix = (int)floor((particle.position.x + halfX) / cellSize);
  int iy = (int)floor((particle.position.y + halfY) / cellSize);
  int iz = (int)floor((particle.position.z + halfZ) / cellSize);

  ix = min(max(ix, 0), gridDimX - 1);
  iy = min(max(iy, 0), gridDimY - 1);
  iz = min(max(iz, 0), gridDimZ - 1);

  particle.cellId = ix + iy * gridDimX + iz * gridDimX * gridDimY;
}

__global__ void initCells(int *cellStart, int *cellEnd, int totalCells) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= totalCells) return;
  cellStart[idx] = -1;
  cellEnd[idx] = -1;
}

// cellStart[i] = the first particle in cell i; cellEnd[i] = the first particle in cell i+1
__global__ void findCellStartEnd(Particle *particles, int particlesCount, int *cellStart, int *cellEnd,
                                 int totalCells) {
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if (idx >= particlesCount) return;
  if (particlesCount == 0) return;
  if (idx == 0) {
    cellStart[particles[0].cellId] = 0;
  } else {
    int cid = particles[idx].cellId;
    int prevCid = particles[idx - 1].cellId;
    if (cid != prevCid) {
      // as particles are sorted, no data race would happen
      cellEnd[prevCid] = idx;
      cellStart[cid] = idx;
    }
  }
  if (idx == particlesCount - 1) {
    cellEnd[particles[particlesCount - 1].cellId] = particlesCount;
  }
}

// Self-defined comparator for thrust::sort
struct ParticleComparator {
  __host__ __device__ bool operator()(const Particle &a, const Particle &b) const { return a.cellId < b.cellId; }
};

// Update: each thread loops only over particles in its own and neighboring grid cells.
__global__ void computeDensityPressureSorted(Particle *particles, int particlesCount, float mass, int *cellStart,
                                             int *cellEnd, float cellSize, int gridDimX, int gridDimY, int gridDimZ,
                                             float xLen, float yLen, float zLen) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= particlesCount) return;
  Particle &particle = particles[i];

  particle.density = 0.0f;
  float h2 = KERNEL_RADIUS * KERNEL_RADIUS;
  float C = mass * POLY6;

  float halfX = xLen / 2.0f;
  float halfY = yLen / 2.0f;
  float halfZ = zLen / 2.0f;
  int ix = min(max((int)floor((particle.position.x + halfX) / cellSize), 0), gridDimX - 1);
  int iy = min(max((int)floor((particle.position.y + halfY) / cellSize), 0), gridDimY - 1);
  int iz = min(max((int)floor((particle.position.z + halfZ) / cellSize), 0), gridDimZ - 1);

  // Loop over neighboring cells (3×3×3)
  for (int dx = -1; dx <= 1; dx++) {
    for (int dy = -1; dy <= 1; dy++) {
      for (int dz = -1; dz <= 1; dz++) {
        int nx = ix + dx;
        int ny = iy + dy;
        int nz = iz + dz;
        if (nx < 0 || nx >= gridDimX || ny < 0 || ny >= gridDimY || nz < 0 || nz >= gridDimZ) continue;
        int neighborCell = nx + ny * gridDimX + nz * gridDimX * gridDimY;
        int start = cellStart[neighborCell];
        int end = cellEnd[neighborCell];
        if (start == -1) continue;
        for (int j = start; j < end; j++) {
          if (j == i) continue;
          float dx = particle.position.x - particles[j].position.x;
          float dy = particle.position.y - particles[j].position.y;
          float dz = particle.position.z - particles[j].position.z;
          float r2 = dx * dx + dy * dy + dz * dz;
          float zVal = h2 - r2;
          if (zVal <= 0 || r2 < 1e-12) continue;
          float rho = C * zVal * zVal * zVal;
          particle.density += rho;
        }
      }
    }
  }
  particle.density += mass * WEIGHT_AT_0;  // contributing to the density of itself
  particle.pressure = (pow(particle.density / REST_DENSITY, 7) - 1.0f) * STIFFNESS;
}

// Update: each thread loops only over particles in its own and neighboring grid cells.
__global__ void computeAccelSorted(Particle *particles, int particlesCount, float mass, int *cellStart, int *cellEnd,
                                   float cellSize, int gridDimX, int gridDimY, int gridDimZ, float xLen, float yLen,
                                   float zLen) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= particlesCount) return;
  Particle &particle = particles[i];

  particle.acceleration.x = 0.0f;
  particle.acceleration.y = 0.0f;
  particle.acceleration.z = 0.0f;

  float h2 = KERNEL_RADIUS * KERNEL_RADIUS;

  float halfX = xLen / 2.0f;
  float halfY = yLen / 2.0f;
  float halfZ = zLen / 2.0f;
  int ix = min(max((int)floor((particle.position.x + halfX) / cellSize), 0), gridDimX - 1);
  int iy = min(max((int)floor((particle.position.y + halfY) / cellSize), 0), gridDimY - 1);
  int iz = min(max((int)floor((particle.position.z + halfZ) / cellSize), 0), gridDimZ - 1);

  // Loop over neighboring cells (3×3×3)
  for (int dx = -1; dx <= 1; dx++) {
    for (int dy = -1; dy <= 1; dy++) {
      for (int dz = -1; dz <= 1; dz++) {
        int nx = ix + dx;
        int ny = iy + dy;
        int nz = iz + dz;
        if (nx < 0 || nx >= gridDimX || ny < 0 || ny >= gridDimY || nz < 0 || nz >= gridDimZ) continue;
        int neighborCell = nx + ny * gridDimX + nz * gridDimX * gridDimY;
        int start = cellStart[neighborCell];
        int end = cellEnd[neighborCell];
        if (start == -1) continue;
        for (int j = start; j < end; j++) {
          if (j == i) continue;
          float dx = particles[i].position.x - particles[j].position.x;
          float dy = particles[i].position.y - particles[j].position.y;
          float dz = particles[i].position.z - particles[j].position.z;
          float r2 = dx * dx + dy * dy + dz * dz;

          if (r2 >= h2 || r2 <= 1e-12) continue;
          float r = sqrtf(r2);

          // pressure force push particles away
          float V = mass / particles[j].density / 2.0f;
          float Kr = KERNEL_RADIUS - r;
          float Kp = SPIKY_GRAD * Kr * Kr;
          float pressureForce = V * (particle.pressure + particles[j].pressure) * Kp;
          particle.acceleration.x -= dx * pressureForce / r;
          particle.acceleration.y -= dy * pressureForce / r;
          particle.acceleration.z -= dz * pressureForce / r;

          // viscosity force pulls particles closer
          float Kv = VISCOSITY_LAPLACIAN * (KERNEL_RADIUS - r);
          float viscosityForce = V * VISCOSITY * Kv;
          float dvx = particles[j].averageVelocity.x - particle.averageVelocity.x;
          float dvy = particles[j].averageVelocity.y - particle.averageVelocity.y;
          float dvz = particles[j].averageVelocity.z - particle.averageVelocity.z;
          particle.acceleration.x += dvx * viscosityForce;
          particle.acceleration.y += dvy * viscosityForce;
          particle.acceleration.z += dvz * viscosityForce;
        }
      }
    }
  }
  particle.acceleration.x /= particle.density;
  particle.acceleration.y /= particle.density;
  particle.acceleration.z /= particle.density;
}

float *allocateMatOnGPU(Mat4 &mat) {
  float data[16];
  int count = 0;
  for (int i = 0; i < 4; i++) {
    for (int j = 0; j < 4; j++) {
      data[count++] = mat[i][j];
    }
  }
  float *matOnGPU;
  hipMalloc((void **)&matOnGPU, 16 * sizeof(float));
  hipMemcpy(matOnGPU, data, 16 * sizeof(float), hipMemcpyHostToDevice);
  return matOnGPU;
}

// Sort particles based on their cell IDs and find the start and end indices of each cell.
void sortParticles(Particle *particles, int particlesCount, int *&cellStart, int *&cellEnd, float cellSize, float xLen,
                   float yLen, float zLen, int gridDimX, int gridDimY, int gridDimZ) {
  int totalCells = gridDimX * gridDimY * gridDimZ;

  int threads = 128;
  int blocks = (particlesCount + threads - 1) / threads;
  computeCellId<<<blocks, threads>>>(particles, particlesCount, cellSize, xLen, yLen, zLen, gridDimX, gridDimY,
                                     gridDimZ);
  hipDeviceSynchronize();

  thrust::device_ptr<Particle> dev_ptr(particles);
  thrust::sort(dev_ptr, dev_ptr + particlesCount, ParticleComparator());

  initCells<<<(totalCells + threads - 1) / threads, threads>>>(cellStart, cellEnd, totalCells);
  hipDeviceSynchronize();
  findCellStartEnd<<<(particlesCount + (threads - 1)) / threads, threads>>>(particles, particlesCount, cellStart,
                                                                            cellEnd, totalCells);
  hipDeviceSynchronize();
}

// Normalize mass based on the density of particles in the sink
float normalizeMass(Particle *particles, int particlesCount, const Sink &sink, int *cellStart, int *cellEnd) {
  float mass = 1.0f;
  int blockDim = 32;
  int gridDim = (particlesCount + (blockDim - 1)) / blockDim;

  // Use Sink dimensions as simulation domain
  float xLen = sink.xLen;
  float yLen = sink.yLen;
  float zLen = sink.zLen;
  float cellSize = KERNEL_RADIUS;
  int gridDimX = (int)ceil(sink.xLen / cellSize);
  int gridDimY = (int)ceil(sink.yLen / cellSize);
  int gridDimZ = (int)ceil(sink.zLen / cellSize);
  sortParticles(particles, particlesCount, cellStart, cellEnd, cellSize, xLen, yLen, zLen, gridDimX, gridDimY,
                gridDimZ);

  computeDensityPressureSorted<<<gridDim, blockDim>>>(particles, particlesCount, mass, cellStart, cellEnd, cellSize,
                                                      gridDimX, gridDimY, gridDimZ, xLen, yLen, zLen);
  hipDeviceSynchronize();

  hipError_t err;
  if ((err = hipGetLastError()) != hipSuccess)
    std::cerr << "Kernel error (computeDensityPressureSorted): " << hipGetErrorString(err) << std::endl;

  float rho0 = REST_DENSITY;
  float rho2s = 0.0f;
  float rhos = 0.0f;
  for (int i = 0; i < particlesCount; i++) {
    rho2s += particles[i].density * particles[i].density;
    rhos += particles[i].density;
  }
  mass *= (rho0 * rhos / rho2s);
  std::cout << "Mass: " << mass << std::endl;

  hipFree(cellStart);
  hipFree(cellEnd);
  return mass;
}

Particle *placeParticles(int &particlesCount, int &droppingParticlesCount, Sink &sink, Trough &trough) {
  float h = KERNEL_RADIUS;
  float hh = h / 2.0f;
  std::cout << "hh: " << hh << std::endl;
  int particlesInSink = 0;
  for (float x = -sink.xLen / 2.0f; x <= sink.xLen / 2.0f; x += hh) {
    for (float z = -sink.zLen / 2.0f; z <= sink.zLen / 2.0f; z += hh) {
      for (float y = -sink.yLen / 2.0f; y <= sink.yLen / 4.0f; y += hh) {
        particlesInSink++;
      }
    }
  }
  int dropping = 0;
  for (float x = trough.vertices[0].x + 0.001f; x <= trough.vertices[1].x - 0.001f; x += hh) {
    float y0 = trough.slope * x + trough.intercept;
    for (float z = -trough.zLen / 2.0f + 0.001f; z <= trough.zLen / 2.0f - 0.001f; z += hh) {
      for (float y = y0; y <= y0 + trough.yLen; y += hh) {
        dropping++;
      }
    }
  }
  droppingParticlesCount = dropping;
  std::cout << "dropping particle count: " << droppingParticlesCount << std::endl;
  particlesCount = particlesInSink + droppingParticlesCount;
  std::cout << "Particle Count: " << particlesCount << std::endl;
  Particle *particles;
  hipMallocManaged(&particles, particlesCount * sizeof(Particle));
  int count = 0;
  // particles generated in sink
  for (float x = -sink.xLen / 2.0f; x <= sink.xLen / 2.0f; x += hh) {
    for (float z = -sink.zLen / 2.0f; z <= sink.zLen / 2.0f; z += hh) {
      for (float y = -sink.yLen / 2.0f; y <= sink.yLen / 4.0f; y += hh) {
        particles[count].position = {x, y, z};
        particles[count].density = 0.0f;
        particles[count].inSink = true;
        particles[count].velocity = {0.0f, 0.0f, 0.0f};
        particles[count].averageVelocity = {0.0f, 0.0f, 0.0f};
        particles[count].acceleration = {0.0f, 0.0f, 0.0f};
        count++;
      }
    }
  }
  // particles that will fall on trough
  float vx = 2.5f;
  float vy = vx * trough.slope;
  for (float x = trough.vertices[0].x + 0.001f; x <= trough.vertices[1].x - 0.001f; x += hh) {
    float y0 = trough.slope * x + trough.intercept;
    for (float z = -trough.zLen / 2.0f + 0.001f; z <= trough.zLen / 2.0f - 0.001f; z += hh) {
      for (float y = y0; y <= y0 + trough.yLen; y += hh) {
        particles[count].position = {x, y, z};
        particles[count].density = 0.0f;
        particles[count].inSink = false;
        particles[count].velocity = {vx, vy, 0.0f};
        particles[count].averageVelocity = {vx, vy, 0.0f};
        particles[count].acceleration = {0.0f, 0.0f, 0.0f};
        count++;
      }
    }
  }
  assert(count == particlesCount);
  return particles;
}

int *initCellStart(int totalCells) {
  int *cellStart;
  hipMalloc(&cellStart, totalCells * sizeof(int));
  return cellStart;
}

int *initCellEnd(int totalCells) {
  int *cellEnd;
  hipMalloc(&cellEnd, totalCells * sizeof(int));
  return cellEnd;
}

Particle *initParticles(int &particlesCount, float &mass, Sink &sink, Trough &trough, int *cellStart, int *cellEnd) {
  int droppingParticlesCount = 0;
  Particle *particles = placeParticles(particlesCount, droppingParticlesCount, sink, trough);
  mass = normalizeMass(particles, particlesCount - droppingParticlesCount, sink, cellStart, cellEnd);
  return particles;
}

__device__ void reflectInSink(Particle &particle, float xLen, float yLen, float zLen) {
  float tbounce = 0.0f;
  if (particle.velocity.x != 0 && (particle.position.x > xLen / 2 || particle.position.x < -xLen / 2)) {
    if (particle.position.x > xLen / 2) {
      tbounce = (particle.position.x - xLen / 2) / particle.velocity.x;
      particle.position.x = xLen - particle.position.x;
    } else {
      tbounce = (particle.position.x + xLen / 2) / particle.velocity.x;
      particle.position.x = -xLen - particle.position.x;
    }
    // revert the movement for the period
    particle.position.y -= particle.velocity.y * (1 - REFLECT_DAMP) * tbounce;
    particle.position.z -= particle.velocity.z * (1 - REFLECT_DAMP) * tbounce;
    particle.velocity.x = -particle.velocity.x;
    particle.velocity.x *= REFLECT_DAMP;
    particle.velocity.y *= REFLECT_DAMP;
    particle.velocity.z *= REFLECT_DAMP;
  }
  if (particle.velocity.y != 0 && (particle.position.y > yLen / 2 || particle.position.y < -yLen / 2)) {
    // bounce back
    if (particle.position.y > yLen / 2) {
      tbounce = (particle.position.y - yLen / 2) / particle.velocity.y;
      particle.position.y = yLen - particle.position.y;
    } else {
      tbounce = (particle.position.y + yLen / 2) / particle.velocity.y;
      particle.position.y = -yLen - particle.position.y;
    }
    // revert the movement for the period
    particle.position.x -= particle.velocity.x * (1 - REFLECT_DAMP) * tbounce;
    particle.position.z -= particle.velocity.z * (1 - REFLECT_DAMP) * tbounce;
    particle.velocity.y = -particle.velocity.y;
    particle.velocity.x *= REFLECT_DAMP;
    particle.velocity.y *= REFLECT_DAMP;
    particle.velocity.z *= REFLECT_DAMP;
  }
  if (particle.velocity.z != 0 && (particle.position.z > zLen / 2 || particle.position.z < -zLen / 2)) {
    // bounce back
    if (particle.position.z > zLen / 2) {
      tbounce = (particle.position.z - zLen / 2) / particle.velocity.z;
      particle.position.z = zLen - particle.position.z;
    } else {
      tbounce = (particle.position.z + zLen / 2) / particle.velocity.z;
      particle.position.z = -zLen - particle.position.z;
    }
    // revert the movement for the period
    particle.position.x -= particle.velocity.x * (1 - REFLECT_DAMP) * tbounce;
    particle.position.y -= particle.velocity.y * (1 - REFLECT_DAMP) * tbounce;
    particle.velocity.z = -particle.velocity.z;
    particle.velocity.x *= REFLECT_DAMP;
    particle.velocity.y *= REFLECT_DAMP;
    particle.velocity.z *= REFLECT_DAMP;
  }
}

__device__ void reflectInTrough(Particle &particle, float zLen, float slope, float intercept, Vec3 normal) {
  float y = particle.position.x * slope + intercept;
  if (y > particle.position.y) {
    // hitting the bottom of the trough: v' = v - 2(v·N)N
    float dotV = particle.velocity.x * normal.x + particle.velocity.y * normal.y + particle.velocity.z * normal.z;
    float newVx = particle.velocity.x - 2 * dotV * normal.x;
    float newVy = particle.velocity.y - 2 * dotV * normal.y;
    float newVz = particle.velocity.z - 2 * dotV * normal.z;
    newVx *= REFLECT_DAMP;
    newVy *= REFLECT_DAMP;
    newVz *= REFLECT_DAMP;
    particle.velocity.x = newVx;
    particle.velocity.y = newVy;
    particle.velocity.z = newVz;
    particle.position.y = y + 0.001;  // simple method putting the particle back on trough
  }
  if (particle.velocity.z != 0 && (particle.position.z > zLen / 2 || particle.position.z < -zLen / 2)) {
    // hitting the side of the trough
    float tbounce = 0.0f;
    if (particle.position.z > zLen / 2) {
      tbounce = (particle.position.z - zLen / 2) / particle.velocity.z;
      particle.position.z = zLen - particle.position.z;
    } else {
      tbounce = (particle.position.z + zLen / 2) / particle.velocity.z;
      particle.position.z = -zLen - particle.position.z;
    }
    // revert the movement for the period
    particle.position.x -= particle.velocity.x * (1 - REFLECT_DAMP) * tbounce;
    particle.position.y -= particle.velocity.y * (1 - REFLECT_DAMP) * tbounce;
    particle.velocity.z = -particle.velocity.z;
    particle.velocity.x *= REFLECT_DAMP;
    particle.velocity.y *= REFLECT_DAMP;
    particle.velocity.z *= REFLECT_DAMP;
  }
}

__global__ void integration(Particle *particles, int particlesCount, float sinkXLen, float sinkYLen, float sinkZLen,
                            float troughZLen, float slope, float intercept, Vec3 normal) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= particlesCount) {
    return;
  }
  Particle &particle = particles[i];
  particle.velocity.x += particle.acceleration.x * DELTA_T;
  particle.velocity.y += particle.acceleration.y * DELTA_T + GRAVITY * DELTA_T;
  particle.velocity.z += particle.acceleration.z * DELTA_T;
  particle.position.x += particle.velocity.x * DELTA_T;
  particle.position.y += particle.velocity.y * DELTA_T;
  particle.position.z += particle.velocity.z * DELTA_T;
  if (particle.inSink == false && (particle.position.x > -sinkXLen / 2.0f && particle.position.x < sinkXLen / 2.0f) &&
      (particle.position.y > -sinkYLen / 2.0f && particle.position.y < sinkYLen / 2.0f) &&
      (particle.position.z > -sinkZLen / 2.0f && particle.position.z < sinkZLen / 2.0f)) {
    particle.inSink = true;
  }
  if (particle.inSink) {
    reflectInSink(particle, sinkXLen, sinkYLen, sinkZLen);
  } else {
    reflectInTrough(particle, troughZLen, slope, intercept, normal);
  }
  particle.averageVelocity.x = (particle.averageVelocity.x + particle.velocity.x) / 2.0f;
  particle.averageVelocity.y = (particle.averageVelocity.y + particle.velocity.y) / 2.0f;
  particle.averageVelocity.z = (particle.averageVelocity.z + particle.velocity.z) / 2.0f;
}

__global__ void coordTransform(Particle *particles, int particlesCount, float *transformMat) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= particlesCount) {
    return;
  }
  Particle &particle = particles[i];
  float worldPos[4], result[4];
  worldPos[0] = particle.position.x;
  worldPos[1] = particle.position.y;
  worldPos[2] = particle.position.z;
  worldPos[3] = 1.0f;
  for (int i = 0; i < 4; i++) {
    result[i] = 0.0f;
    for (int j = 0; j < 4; j++) {
      result[i] += transformMat[i * 4 + j] * worldPos[j];
    }
  }
  float x = result[0] / result[3];
  float y = result[1] / result[3];

  if (x < -1.0f || x > 1.0f || y < -1.0f || y > 1.0f) {
    particle.screenPos.x = -1.0f;
    particle.screenPos.y = -1.0f;
  } else {
    float screenX = fmaxf(0.0f, fminf(1.0f, (x + 1.0f) * 0.5f)) * SCREEN_WIDTH;
    float screenY = fmaxf(0.0f, fminf(1.0f, (1.0f - y) * 0.5f)) * SCREEN_HEIGHT;
    particle.screenPos.x = screenX;
    particle.screenPos.y = screenY;
  }
}

void updateSimulation(Particle *particles, int particlesCount, const Sink &sink, const Trough &trough, float mass,
                      float *transformMat, int *cellStart, int *cellEnd) {
  int blockDim = 32;
  int gridDim = (particlesCount + (blockDim - 1)) / blockDim;

  float cellSize = KERNEL_RADIUS;
  float xLen = sink.xLen;
  float yLen = sink.yLen;
  float zLen = sink.zLen;
  int gridDimX = (int)ceil(xLen / cellSize);
  int gridDimY = (int)ceil(yLen / cellSize);
  int gridDimZ = (int)ceil(zLen / cellSize);

  sortParticles(particles, particlesCount, cellStart, cellEnd, cellSize, xLen, yLen, zLen, gridDimX, gridDimY,
                gridDimZ);

  hipError_t err;
  computeDensityPressureSorted<<<gridDim, blockDim>>>(particles, particlesCount, mass, cellStart, cellEnd, cellSize,
                                                      gridDimX, gridDimY, gridDimZ, xLen, yLen, zLen);
  hipDeviceSynchronize();
  if ((err = hipGetLastError()) != hipSuccess)
    std::cerr << "Kernel error (computeDensityPressureSorted): " << hipGetErrorString(err) << std::endl;

  computeAccelSorted<<<gridDim, blockDim>>>(particles, particlesCount, mass, cellStart, cellEnd, cellSize, gridDimX,
                                            gridDimY, gridDimZ, xLen, yLen, zLen);
  if ((err = hipGetLastError()) != hipSuccess)
    std::cerr << "Kernel error (computeAccelSorted): " << hipGetErrorString(err) << std::endl;

  integration<<<gridDim, blockDim>>>(particles, particlesCount, sink.xLen, sink.yLen, sink.zLen, trough.zLen,
                                     trough.slope, trough.intercept, trough.normal);
  coordTransform<<<gridDim, blockDim>>>(particles, particlesCount, transformMat);
  hipDeviceSynchronize();
  if ((err = hipGetLastError()) != hipSuccess)
    std::cerr << "Kernel error (integration or coordTransform): " << hipGetErrorString(err) << std::endl;

  hipFree(cellStart);
  hipFree(cellEnd);
}
